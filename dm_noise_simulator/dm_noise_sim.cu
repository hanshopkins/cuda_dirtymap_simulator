#include "hip/hip_runtime.h"
//compile line: nvcc --shared -o gpu_dmns.so --compiler-options -fPIC dm_noise_sim.cu
#include <iostream>
#include <hiprand/hiprand_kernel.h>
#include <math.h> //sqrt

#define PI 3.14159265358979323846f
#define PIXELS_PER_BLOCK 128

typedef struct
{
	float x;
	float y;
	float z;
} vector3;

class myComplex
{
	public:
		float real;
		float imag;
	__global__ myComplex(float real_, float imag_)
	{
		real = real_;
		imag = imag_;
	}
	myComplex& operator+= (const myComplex& rhs)
	{
		real += rhs.real;
		imag += rhs.imag;
		return *this;
	}
};

__global__ inline myComplex operator* (const float a, const myComplex b) {return myComplex(b.real*a, b.imag*a)};
__global__ inline myComplex operator* (const myComplex b, const float a) {return myComplex(b.real*a, b.imag*a)};

__device__ exp (myComplex x)
{
	return expf(x.real)*myComplex(sinf(x.imag),cosf(x.imag));
}

__global__ void fill_noise_draws (myComplex* noise_draws, int nbaselines, int ntimesamples, float* stdv, unsigned long long freq_idx, unsigned long long seed)
{
	int idx = threadIdx.x + blockIdx.x *32;
	hiprandStateMRG32k3a_t state;
	hiprand_init(seed, idx+freq_idx*nbaselines*ntimesamples, 0, &state);
	if (idx < nbaselines*ntimesamples)
	{
		noise_draws[idx] = myComplex(hiprand_normal(&state), hiprand_normal(&state))*stdv[idx/(ntimesamples*2)];
	}
}

inline vector3 ang2vec (const float theta, const float phi)
{
	return {sin(theta)*cos(phi), sin(theta)*sin(phi), cos(theta)};
}

inline vector3 cross (const vector3 v1, const vector3 v2)
{
	return {v1.y*v2.z-v1.z*v2.y,
			v1.z*v2.x-v1.x*v2.z,
			v1.x*v2.y-v1.y*v2.x};
}

__device__ inline vector3 rotate (const vector3 v, const float alpha)
{
	return {cosf(alpha)*v.x - sinf(alpha)*v.y, sinf(alpha)*v.x + cosf(alpha)*v.y, v.z};
}

__device__ inline float dot (const vector3 v1, const vector3 v2)
{
    return v1.x*v2.x+v1.y*v2.y+v1.z*v2.z;
}

__device__ inline float angular_difference (const vector3 v1, const vector3 v2)
{
	vector3 c = cross(v1, v2);
	return atan2f(sqrtf(dot(c,c)), dot(v1,v2));
}

__device__ float B_sq (const float alpha, const float wavelength, const float D)
{
    float alphaprime = PI*D*sinf(alpha)/wavelength;
    if (alphaprime <= 1E-8f && alphaprime >= -1E-8f)
        return (j0f(alphaprime)-jnf(2,alphaprime))*(j0f(alphaprime)-jnf(2,alphaprime)); //l'Hopital's
    else
        return (2*j1f(alphaprime)/alphaprime) * (2*j1f(alphaprime)/alphaprime);
}

__global__ void dm_noise_sim (const myComplex<float>* visibility_noise_draws, float* stdv,
    const vector3* u, const vector3* baselines, int nbaselines, float wavelength,
    vector3 telescope_u, float dish_diameter, float deg_distance_to_count, int ntimesamples_full, int ntimesamples, float* noise_map)
{   
	int pixelidx = threadIdx.x + blockIdx.x*PIXELS_PER_BLOCK;
    float pixelphi = atan2f(u[pixelidx].x, u[pixelidx].y);
    int rough_time_placement = pixelphi/(2*PI) * ntimesamples_full;
    int t_initial = rough_time_placement-ntimesamples/2;
    myComplex sum = 0;
    for (int i = 0; i < nbaselines; i++)
    {
    	float inv_cov = 1.0/(stdv[i]*stdv[i]);
        for (int t = t_initial; t < t_initial+ntimesamples; t++)
        {
            float delta_phi = (2*PI/ntimesamples_full)*t;
            vector3 u_rot = rotate(u[pixelidx], delta_phi);
            float Bsq = B_sq(angular_difference(u_rot, telescope_u), wavelength, dish_diameter);
            sum += inv_cov*visibility_noise_draws[i*ntimesamples+t]*exp(myComplex(0,-2*PI/wavelength*dot(baselines[i],u_rot)));
        }
    }
    noise_map[pixelidx] = sum.real;
}

extern "C" {void dm_noise_sim_caller (float noise,
    const vector3* u, unsigned int npixels, const float* baselines, const int* baseline_counts, int nbaselines, const float* wavelengths, int nwavelengths,
    float telescope_dec, float dish_diameter, float deg_distance_to_count, int ntimesamples_full, float* noise_map, unsigned long long seed)
{
    float telescope_theta = (90-telescope_dec)*PI/180;

    //making unit vector baselines
    vector3 zenith_basis = ang2vec(telescope_theta,0);
    vector3 ns_basis = ang2vec(telescope_theta-PI/2,0);
    vector3 ew_basis = cross(ns_basis,zenith_basis);

    vector3* a = new vector3 [nbaselines];
    float* stdv = new float [nbaselines]; //setting up list of standard deviations for each baseline
    for (int i = 0; i < nbaselines; i++)
    {
        a[i] = {baselines[2*i]*ew_basis.x + baselines[2*i+1]*ns_basis.x,
         		baselines[2*i]*ew_basis.y + baselines[2*i+1]*ns_basis.y,
         		baselines[2*i]*ew_basis.z + baselines[2*i+1]*ns_basis.z};
	    if (baselines[2*i] == 0 && baselines[2*i+1] == 0)
	    {
	        stdv[i] = sqrt(baseline_counts[i])*noise * sqrt(2)/2;
	    }
	    else
	        stdv[i] = sqrt(baseline_counts[i])*noise;
    }

	//we don't want to actually count every time step, since the majority of them are 0 because of B_sq
    int ntimesamples = int(ntimesamples_full * (2*deg_distance_to_count)/360.0);

	//setting up multiple GPUs and cuda stuff
	int deviceCount;
    hipGetDeviceCount(&deviceCount);
	std::cout << "Device count: " << deviceCount << std::endl;
	
	//padding
	unsigned int npixelblocks_padded = (npixels+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK;
	int npixels_padded = npixelblocks_padded*PIXELS_PER_BLOCK;
	unsigned int gpu_pixel_idx[deviceCount]; //where each GPU starts considering pixels
	unsigned int gpu_pixel_length[deviceCount]; //how many pixels the GPU computes
	for (int gpuId = 0; gpuId < deviceCount; gpuId++)
	{
		unsigned int pixelblock_idx = npixelblocks_padded/deviceCount * gpuId;
		gpu_pixel_idx[gpuId] = pixelblock_idx * PIXELS_PER_BLOCK;
		unsigned int pixelblock_length = gpuId + 1 == deviceCount ? npixelblocks_padded - pixelblock_idx : npixelblocks_padded/deviceCount;
		gpu_pixel_length[gpuId] = pixelblock_length * PIXELS_PER_BLOCK;
	}
	vector3* u_padded = new vector3 [npixels_padded];
	for (unsigned int i = 0; i<npixels; i++) u_padded[i] = u[i];
	for (unsigned int i = npixels; i < npixels_padded; i++)
    {
    	u_padded[i] = {1,0,0};
    }
    vector3* u_d[deviceCount];
    
	float* padded_noise_map_d[deviceCount];
	myComplex* noise_draws[deviceCount];
	vector3* a_d[deviceCount];
	float* stdv_d[deviceCount];
	for (int gpuId = 0; gpuId < deviceCount; gpuId++)
	{
		hipSetDevice(gpuId);
		hipMalloc(&u_d[gpuId], sizeof(vector3)*gpu_pixel_length[gpuId]);
		hipMemcpyAsync(u_d[gpuId], u + gpu_pixel_idx[gpuId], sizeof(vector3)*gpu_pixel_length[gpuId], hipMemcpyHostToDevice);
		hipMalloc(&padded_noise_map_d[gpuId], sizeof(float)*gpu_pixel_length[gpuId]);
		hipMalloc(&noise_draws[gpuId], sizeof(myComplex)*nbaselines*ntimesamples_full);
		hipMalloc(&a_d[gpuId], sizeof(vector3)*nbaselines);
		hipMemcpyAsync(a_d[gpuId], a, sizeof(vector3)*nbaselines, hipMemcpyHostToDevice);
		hipMalloc(&stdv_d[gpuId], sizeof(float)*nbaselines);
		hipMemcpyAsync(stdv_d[gpuId], stdv, sizeof(float)*nbaselines, hipMemcpyHostToDevice);
	}
	delete[] u_padded;
	delete[] a;
	delete[] stdv;
	float* padded_noise_map = new float [npixels_padded*nwavelengths];
	for (int l = 0; l < nwavelengths; l++)
	{	
		for (int gpuId = 0; gpuId < deviceCount; gpuId++)
		{
			hipSetDevice(gpuId);
			int nblocks = (2*nbaselines*ntimesamples_full + 31)/32;
			fill_noise_draws <<<nblocks, 32>>> (noise_draws[gpuId], nbaselines, ntimesamples_full, stdv_d[gpuId], l, seed);
		}
		hipDeviceSynchronize();
		for (int gpuId = 0; gpuId < deviceCount; gpuId++)
		{
			hipSetDevice(gpuId);
			int nblocks = gpu_pixel_length[gpuId]/PIXELS_PER_BLOCK;
			dm_noise_sim<<<nblocks,PIXELS_PER_BLOCK>>> (noise_draws[gpuId], stdv_d[gpuId],
    			u_d[gpuId], a_d[gpuId], nbaselines, wavelengths[l],
    			zenith_basis, dish_diameter, deg_distance_to_count, ntimesamples_full, ntimesamples, padded_noise_map_d[gpuId]);
		}
		hipDeviceSynchronize();
		//note the way this is written, it's nwavelengths*npixels instead of npixels*nwavelengths like the other code. Maybe I can fix this while unpadding.
		for (int gpuId = 0; gpuId <  deviceCount; gpuId++)
		{
			hipMemcpyAsync(padded_noise_map+l*npixels_padded + gpu_pixel_idx[gpuId], padded_noise_map_d[gpuId], sizeof(float)*gpu_pixel_length[gpuId], hipMemcpyDeviceToHost);
		}
		hipDeviceSynchronize();
	}
	
	//last we just do unpadding and transpose on the CPU
	for (unsigned int i = 0; i<npixels; i++)
	{
		for (unsigned int l = 0; l<nwavelengths; l++)
		{
			noise_map[i*nwavelengths+l] = padded_noise_map[l*npixels_padded+i];
		}
	}
	delete[] padded_noise_map;
	for (int gpuId = 0; gpuId < deviceCount; gpuId++)
	{
		hipSetDevice(gpuId);
		hipFree(u_d[gpuId]);
		hipFree(padded_noise_map_d[gpuId]);
		hipFree(noise_draws[gpuId]);
		hipFree(a_d[gpuId]);
	}
}
}
