//compiling line: nvcc --shared -o dms.so --compiler-options -fPIC dirty_map.cu
#include <hip/hip_runtime.h>
#include <math.h>       /* sin, cos, fmod, fabs, asin, atan2 */
#include <stdio.h>
#include <iostream>

#define PI 3.14159265358979323846
#define omega 2*PI/86400 //earth angular velocity in rads/second

struct floatArray {
    float * p;
    unsigned int l;
};

struct chordParams
{
    floatArray thetas;
    float initial_phi_offset; //amount that the calculation starts away from each source
    unsigned int m1; //north south number of dishes
    unsigned int m2; //east west
    float L1; // north osuth dish separation
    float L2; //east west
    float CHORD_zenith_dec;
    float D; //dish diameter
    float delta_tau;
    unsigned int time_samples;
};

__device__ inline void ang2vec (const float theta, const float phi, float outvec [3])
{
    outvec[0] = sin(theta)*cos(phi);
    outvec[1] = sin(theta)*sin(phi);
    outvec[2] = cos(theta);
}

__device__ inline void cross (const float v1 [3], const float v2 [3], float outvec [3])
{
    outvec[0] = v1[1]*v2[2]-v1[2]*v2[1];
    outvec[1] = v1[2]*v2[0]-v1[0]*v2[2];
    outvec[2] = v1[0]*v2[1]-v1[1]*v2[0]; 
}

__device__ inline void rotate (const float v [3], float outvec [3], const float alpha)
{
    outvec[0] = cos(alpha)*v[0] - sin(alpha)*v[1];
    outvec[1] = sin(alpha)*v[0] + cos(alpha)*v[1];
    outvec[2] = v[2];
}

__device__ inline float dot (const float v1 [3], const float v2 [3])
{
    return v1[0]*v2[0]+v1[1]*v2[1]+v1[2]*v2[2];
}

__device__ inline float crossmag(const float v1 [3], const float v2 [3])
{
    float cv [3];
    cross(v1,v2,cv);
    return sqrt(dot(cv,cv));
}

__device__ float B_sq (const float alpha, const float wavelength, const float D)
{
    float alphaprime = PI*D*sin(alpha)/wavelength;
    if (alphaprime <= 1E-8 && alphaprime >= -1E-8)
        return (j0f(alphaprime)-jnf(2,alphaprime))*(j0f(alphaprime)-jnf(2,alphaprime)); //l'Hopital's
    else
        return (2*j1f(alphaprime)/alphaprime) * (2*j1f(alphaprime)/alphaprime);
}

__device__ inline float Bsq_from_vecs (const float v1 [3], const float v2 [3], const float wavelength, const float D)
{
    float dp = dot(v1,v2);
    if (dp <= 0) return 0; //horizon condition
    else
    {
        //we want to deal with the arccos instiblity by using the cross product formula instead
        float delta_ang;
        if (dp < 0.99) delta_ang = std::acos(dp);    
        else 
        {
            delta_ang = std::asin(crossmag(v1,v2));
            //delta_ang = (dp > 0) ? delta_ang : PI-delta_ang; //I don't need this line with the horizon condition
        }
        return B_sq(delta_ang, wavelength, D);
    }
}

__device__ inline float subtractdot (const float v1_a [3], const float v1_b [3], const float v2 [3])
{
    return (v1_a[0]-v1_b[0])*v2[0]+(v1_a[1]-v1_b[1])*v2[1]+(v1_a[2]-v1_b[2])*v2[2];
}

__device__ float sin_sq_ratio (const unsigned int m, const float x_prime)
{
    float x = fmodf(x_prime,PI); // -pi < x < pi
    x = fabs(x); // 0 < x < pi
    x = (x > PI/2) ? PI-x : x; //0 < x < pi/2
    
    if (fabs(x) < 1E-9) return m*m*cos(m*x)*cos(m*x)/(cos(x)*cos(x));
    else return sin(m*x)*sin(m*x)/(sin(x)*sin(x));
}

__global__ void dirtymap_kernel (const floatArray u, const floatArray wavelengths, const floatArray source_positions, const floatArray source_spectra, float brightness_threshold, const chordParams cp, float * dm)
{
    if (blockIdx.x*32 + threadIdx.x < u.l / 3)
    {
        //calculating the relevant CHORD vectors for each dither direction
        float * chord_pointing = new float [3*cp.thetas.l];
        float * dir1_proj_vec = new float [3*cp.thetas.l]; //north/south chord direction
        float * dir2_proj_vec = new float [3*cp.thetas.l]; //east/west chord direction
        for (unsigned int k = 0; k < cp.thetas.l; k++)
        {
            ang2vec(cp.thetas.p[k], 0, chord_pointing+3*k);
            ang2vec(cp.thetas.p[k] + PI/2, 0, dir1_proj_vec+3*k);
            cross(dir1_proj_vec+3*k, chord_pointing+3*k, dir2_proj_vec+3*k);
        }
        //accounting for CHORD's baseline shrinking when it points away from zenith
        float * L1s = new float [cp.thetas.l];
        for (unsigned int k = 0; k < cp.thetas.l; k++)
        {
            L1s[k] = cp.L1*cos(PI/180*(90-cp.CHORD_zenith_dec) - cp.thetas.p[k]);
        }

        float * threadu = u.p + blockIdx.x*32 + threadIdx.x;
	if (blockIdx.x*32 + threadIdx.x == u.l/3/2) printf("u: (%f, %f, %f), chord u: (%f,%f,%f)\n", threadu[0], threadu[1], threadu[2], chord_pointing[0], chord_pointing[1], chord_pointing[2]);
        for (unsigned int l = 0; l < wavelengths.l; l++)
        {
            float usum = 0;
            for (unsigned int s = 0; s*wavelengths.l < source_spectra.l; s++)
            {
                float time_sum = 0;
                if (source_spectra.p[s*wavelengths.l + l] > brightness_threshold)
                {
		    float source_phi = atan2(source_positions.p[s*3+1],source_positions.p[s*3]);
		    float initial_tau = -(source_phi-cp.initial_phi_offset)/omega; //we want it to start computing phi_offset away from the source
                    for (unsigned int k = 0; k < cp.thetas.l; k++)
                    {
                        for (unsigned int j = 0; j < cp.time_samples; j++)
                        {
                            float tau = initial_tau+j*cp.delta_tau;
                            float u_rot [3];
                            rotate(threadu, u_rot, tau*omega);
                            float source_rot [3];
                            rotate(source_positions.p+3*s, source_rot, tau*omega);

                            float cdir1 = PI*L1s[k]/wavelengths.p[l]*subtractdot(source_rot, u_rot, dir1_proj_vec+3*k);
                            float cdir2 = PI*cp.L2 /wavelengths.p[l]*subtractdot(source_rot, u_rot, dir2_proj_vec+3*k);

                            float Bsq_source = Bsq_from_vecs(source_rot, chord_pointing+3*k, wavelengths.p[l], cp.D);
                            float Bsq_u = Bsq_from_vecs(u_rot, chord_pointing+3*k, wavelengths.p[l], cp.D);

                            time_sum += Bsq_source * Bsq_u * sin_sq_ratio(cp.m1,cdir1) * sin_sq_ratio(cp.m2,cdir2);
			    if (blockIdx.x*32 + threadIdx.x == u.l/3/2) printf("bsq_source, bsqu, and sinsqu parts: %e %e %e %e\n", Bsq_source, Bsq_u, sin_sq_ratio(cp.m1,cdir1), sin_sq_ratio(cp.m2,cdir2));
			    if (blockIdx.x*32 + threadIdx.x == u.l/3/2) printf("Time sum at middle pixel: %e\n", time_sum);
                        }
                    }
                }
                usum += source_spectra.p[s*wavelengths.l + l] * time_sum;
            }
            dm[(blockIdx.x*32 + threadIdx.x)*wavelengths.l + l] = usum;
            if (blockIdx.x*32 + threadIdx.x == u.l/3/2) printf("Total sum at middle pixel: %e\n", usum);
        }
    delete chord_pointing;
    delete dir1_proj_vec;
    delete dir2_proj_vec;
    delete L1s;
    }
}

inline void copyFloatArrayToDevice (const floatArray host_array, floatArray & device_array)
{
    device_array.l = host_array.l;
    hipMalloc(&device_array.p, sizeof(float) * host_array.l);
    hipMemcpy(device_array.p, host_array.p, sizeof(float) * host_array.l, hipMemcpyHostToDevice);
}

extern "C" {void dirtymap_caller(const floatArray u, const floatArray wavelengths, const floatArray source_positions, const floatArray source_spectra, float brightness_threshold, const chordParams cp, float * dm)
{
    unsigned int npixels = u.l/3;
    //copying data over to the device
    floatArray d_u;
    copyFloatArrayToDevice(u,d_u);
    floatArray d_wavelengths;
    copyFloatArrayToDevice(wavelengths,d_wavelengths);
    floatArray d_source_positions;
    copyFloatArrayToDevice(source_positions, d_source_positions);
    floatArray d_source_spectra;
    copyFloatArrayToDevice(source_spectra,d_source_spectra);
    chordParams d_cp = cp;
    floatArray d_thetas;
    copyFloatArrayToDevice(cp.thetas,d_thetas);
    d_cp.thetas = d_thetas;
    float * d_dm;
    hipMalloc(&d_dm, sizeof(float)*npixels*wavelengths.l);

    dirtymap_kernel<<<(npixels+31)/32,32>>>(d_u, d_wavelengths, d_source_positions, d_source_spectra, brightness_threshold, d_cp, d_dm);
    hipMemcpy(dm, d_dm, sizeof(float)*npixels*wavelengths.l, hipMemcpyDeviceToHost);

    hipFree(d_u.p);
    hipFree(d_wavelengths.p);
    hipFree(d_source_positions.p);
    hipFree(d_source_spectra.p);
    hipFree(d_cp.thetas.p);
    hipFree(d_dm);
}
}
